#include "hip/hip_runtime.h"
#include <THC.h>
#include <stdbool.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "back_projection_kernel.h"

const int CUDA_NUM_THREADS = 1024;

// 计算出在给定数据量和线程数的情况下，需要启动多少个CUDA块来处理数据
inline int GET_BLOCKS(const int N){
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

// 在不支持Compute Capability 6.0及以上版本的CUDA设备上提供一个替代方案，以实现原子加法操作
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
__device__ double atomicAdd(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                            __longlong_as_double(assumed)));
  // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif


#define EPS 1e-5
// for rounding to the nearest integer


// 将一个浮点数取整为最接近的小于或等于它的整数（向下取整）
#define FLOOR_I(a) \
  ( (a) < 0 ? (int)(a) - 1:(int) (a) )

// 将一个浮点数取整为最接近的小于或等于它的整数，并将结果转换为浮点数类型
#define FLOOR_F(a)                                              \
  (float)(FLOOR_I(a))

// 将一个浮点数进行四舍五入，并返回最接近的整数值
#define ROUND_I(a) \
  ( (a)-FLOOR_F(a) > FLOOR_F(a) + 1.0 - (a) ? FLOOR_I(a)+1:FLOOR_I(a) )

// 将一个浮点数进行四舍五入，并返回最接近的浮点数值
#define ROUND_F(a)                                              \
  ( (a)-FLOOR_F(a) > FLOOR_F(a) + 1.0 - (a) ? FLOOR_F(a)+1:FLOOR_F(a) )

// 判断给定的三个浮点数x1、x2、x3是否在给定的三个范围s1、s2、s3内
#define WITHIN_BOUNDS(x1, x2, x3, s1, s2, s3) ( (x1 >= 0.0f) && (x1 < s1) && (x2 >= 0.0f) && (x2 < s2) && (x3 >= 0.0f) && (x3 < s3))

// 根据给定的索引和步长从四维数据数组中获取对应的元素值
#define GET_DIRECT_4d(data, x0, x1, x2, x3, sd0, sd1, sd2, sd3)         \
  ((data)[(x0) * (sd0) + (x1) * (sd1) + (x2) * (sd2) + (x3) * (sd3)])

// 在四维数据数组中的特定位置进行原子加法操作
#define ADD_ATOMIC_4d(data, x0, x1, x2, x3, sd0, sd1, sd2, sd3, v)        \
  atomicAdd( data + (x0) * (sd0) + (x1) * (sd1) + (x2) * (sd2) + (x3) * (sd3), v )
// 在五维数据数组中的特定位置进行原子加法操作
#define ADD_ATOMIC_5d(data, x0, x1, x2, x3, x4, sd0, sd1, sd2, sd3, sd4, v) \
  atomicAdd( data + (x0) * (sd0) + (x1) * (sd1) + (x2) * (sd2) + (x3) * (sd3) + (x4)*(sd4), v )
// 在四维数据数组中的特定位置设置一个给定的值
#define SET_DIRECT_4d(data, x0, x1, x2, x3, sd0, sd1, sd2, sd3, v)        \
  ((data)[(x0) * (sd0) + (x1) * (sd1) + (x2) * (sd2) + (x3) * (sd3)]) = v

// 根据给定的索引和步长从三维数据数组中获取对应的元素值
#define GET_DIRECT_3d(data, x0, x1, x2, sd0, sd1, sd2) \
  ((data)[(x0) * (sd0) + (x1) * (sd1) + (x2) * (sd2)])


// 在三维数据数组中的特定位置设置一个给定的值
#define SET_DIRECT_3d(data, x0, x1, x2, sd0, sd1, sd2, v)        \
  ((data)[(x0) * (sd0) + (x1) * (sd1) + (x2) * (sd2) ]) = v


// 根据给定的索引和步长从五维数据数组中获取对应的元素值
#define GET_DIRECT_5d(data, x0, x1, x2, x3, x4, stride0, stride1, stride2, stride3, stride4) \
  ((data)[(x0)*(stride0)+(x1)*(stride1)+(x2)*(stride2)+(x3)*(stride3)+(x4)*(stride4)])


// 在五维数据数组中的特定位置设置一个给定的值
#define SET_DIRECT_5d(data, x0, x1, x2, x3, x4, stride0, stride1, stride2, stride3, stride4, value) \
  ((data)[(x0)*(stride0)+(x1)*(stride1)+(x2)*(stride2)+(x3)*(stride3)+(x4)*(stride4)] = (value))


// 将全局坐标下的体素索引转换为体素坐标，将全局坐标(glob_a)加上0.5后乘以体素分辨率(res_a)，然后取整为最接近的小于或等于的整数，从而得到对应的体素坐标
#define VOXIND_TO_VOXC(glob_a, res_a)           \
  ( FLOOR_I( (glob_a+0.5f) * (float)res_a ) )

// 返回两个值中的较大值
#define MAX(a,b) ( ((a)>(b)) ? (a) : (b) )

// 返回三个值中的最大值
#define MAX3(a,b,c) MAX( MAX((a),(b)), (c) )

// 生成一个用于并行计算的循环结构，在CUDA内核函数中使用
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)


// 进行参数检查和错误处理，用于在条件不满足时抛出异常
#define THCUNN_argCheck(STATE, COND, ARG, T, FORMAT) \
  if (!(COND)) { \
    THCDescBuff s1 = THCudaTensor_sizeDesc(state, T); \
    THArgCheck(COND, ARG, FORMAT, s1.str);           \
  }

// 进行维度和大小的检查，并在条件不满足时抛出异常
#define THCUNN_check_dim_size(STATE, T, DIM, DIM_SIZE, SIZE) \
  if (THCudaTensor_nDimension(STATE, T) != DIM ||             \
      THCudaTensor_size(STATE, T, DIM_SIZE) != SIZE) {        \
      THCDescBuff s1 = THCudaTensor_sizeDesc(state, T);       \
      THError("Need " #T " of dimension %d and " #T ".size[%d] == %d" \
              " but got " #T " to be of shape: %s", DIM, DIM_SIZE, SIZE, s1.str); \
  }

// 确保多个张量在同一个GPU上
#define THCUNN_assertSameGPU(...) THAssertMsg(THCudaTensor_checkGPU(__VA_ARGS__), \
  "Some of weight/gradient/input tensors are located on different GPUs. Please move them to a single one.")




// 进行输入张量的形状检查
static inline void cambp_shapecheck(THCState* state,
                                    THCudaTensor* depth,
                                    THCudaTensor* camdist,
                                    THCudaTensor* fl,
                                    THCudaTensor* voxel,
                                    THCudaTensor* cnt){
  THCUNN_argCheck(state, THCudaTensor_nDimension(state, depth) == 4, 2, depth,
      "4D input tensor expected but got: %s");
  THCUNN_argCheck(state, THCudaTensor_nDimension(state, camdist) == 2, 2, camdist,
      "3D input tensor expected but got: %s");
  THCUNN_argCheck(state, THCudaTensor_nDimension(state, fl) == 2, 2, fl,
      "3D input tensor expected but got: %s");

  THCUNN_argCheck(state, THCudaTensor_nDimension(state, cnt) == 5, 2, cnt,
      "5D input tensor expected but got: %s");

  THCUNN_argCheck(state, THCudaTensor_nDimension(state, voxel) == 5, 2, voxel,
      "5D input tensor expected but got: %s");

  int nbatch = THCudaTensor_size(state, depth, 0);
  int nc = THCudaTensor_size(state, depth, 1);
  int vszx = THCudaTensor_size(state, voxel, 2);
  int vszy = THCudaTensor_size(state, voxel, 3);
  int vszz = THCudaTensor_size(state, voxel, 4);

  //fprintf(stderr, "argcheck + size pass\n");
  THCUNN_check_dim_size(state, camdist, 2, 0, nbatch);
  THCUNN_check_dim_size(state, camdist, 2, 1, nc);
  //fprintf(stderr, "camdist  pass\n");
  THCUNN_check_dim_size(state, fl, 2, 0, nbatch);
  THCUNN_check_dim_size(state, fl, 2, 1, nc);
  THCUNN_check_dim_size(state, voxel, 5, 0, nbatch);
  THCUNN_check_dim_size(state, voxel, 5, 1, nc);
  THCUNN_check_dim_size(state, cnt, 5, 0, nbatch);
  THCUNN_check_dim_size(state, cnt, 5, 1, nc);
  THCUNN_check_dim_size(state, cnt, 5, 2, vszx);
  THCUNN_check_dim_size(state, cnt, 5, 3, vszy);
  THCUNN_check_dim_size(state, cnt, 5, 4, vszz);

}

// 检查输入张量的维度和大小
static inline void cambp_shapecheck(THCState* state,
                                    THCudaTensor* depth,
                                    THCudaTensor* grid,
                                    THCudaTensor* voxel,
                                    THCudaTensor* cnt){
  THCUNN_argCheck(state, THCudaTensor_nDimension(state, depth) == 4, 2, depth,
      "4D input tensor expected but got: %s");
  THCUNN_argCheck(state, THCudaTensor_nDimension(state, grid) == 5, 2, grid,
      "5D input tensor expected but got: %s");

  THCUNN_argCheck(state, THCudaTensor_nDimension(state, cnt) == 5, 2, cnt,
      "5D input tensor expected but got: %s");

  THCUNN_argCheck(state, THCudaTensor_nDimension(state, voxel) == 5, 2, voxel,
      "5D input tensor expected but got: %s");

  int nbatch = THCudaTensor_size(state, depth, 0);
  int nc = THCudaTensor_size(state, depth, 1);
  int nh = THCudaTensor_size(state, depth, 2);
  int nw = THCudaTensor_size(state, depth, 3);
  int vszx = THCudaTensor_size(state, voxel, 2);
  int vszy = THCudaTensor_size(state, voxel, 3);
  int vszz = THCudaTensor_size(state, voxel, 4);

  //fprintf(stderr, "argcheck + size pass\n");
  THCUNN_check_dim_size(state, grid, 5, 0, nbatch);
  THCUNN_check_dim_size(state, grid, 5, 1, nc);
  THCUNN_check_dim_size(state, grid, 5, 2, nh);
  THCUNN_check_dim_size(state, grid, 5, 3, nw);
  THCUNN_check_dim_size(state, grid, 5, 4, 3);
  //fprintf(stderr, "camdist  pass\n");
  THCUNN_check_dim_size(state, voxel, 5, 0, nbatch);
  THCUNN_check_dim_size(state, voxel, 5, 1, nc);
  THCUNN_check_dim_size(state, cnt, 5, 0, nbatch);
  THCUNN_check_dim_size(state, cnt, 5, 1, nc);
  THCUNN_check_dim_size(state, cnt, 5, 2, vszx);
  THCUNN_check_dim_size(state, cnt, 5, 3, vszy);
  THCUNN_check_dim_size(state, cnt, 5, 4, vszz);
}



// 计算给定值的平方
#define square(a) \
  ((a)*(a))

//inline float sqrt(float a){
//  return sqrtf(a);
//}


// 计算三维向量的欧几里德范数（模长）
#define vec3d_norm( x1, x2, x3) \
  sqrtf(square(x1) + square(x2) + square(x3))


// 用于设置CUDA内核函数的启动配置
__launch_bounds__(CUDA_NUM_THREADS)
// 用于执行后向投影的前向操作
__global__ void back_projection_forward_kernel(float* depth,
                                               int N, int NC,
                                               int dszh, int dszw,
                                               int dsdn, int dsdc, int dsdh, int dsdw,
                                               float* cam_dist_in,
                                               int cdisdn, int cdisdc,
                                               float* fl_in,
                                               int fisdn,int fisdc,
                                               float* voxel,
                                               int vszx, int vszy, int vszz,
                                               int vsdn, int vsdc, int vsdx, int vsdy, int vsdz,
                                               float* cnt,
                                               int csdn, int csdc, int csdx, int csdy, int csdz,
                                               int nthreads){
  // int index;
  CUDA_KERNEL_LOOP(index, nthreads){
    const int n = index % N; // 当前线程所处理的 批次 索引
    const int ind_c = (index / N) % NC; // 根据索引index、批次大小N和通道数NC计算当前线程所处理的 通道 索引
    const int ind_w = (index / (N * NC)) % dszh; // 根据索引index、批次大小N和通道数NC以及深度张量高度dszh计算当前线程所处理的 高度 索引
    const int ind_h = (index / (N * NC * dszh)) % dszw; // 当前线程所处理的 宽度 索引

    // 获取深度张量在给定位置(n, ind_c, ind_h, ind_w)处的值，并将其存储在变量dep_at_pix中
    float dep_at_pix = GET_DIRECT_4d(depth, n, ind_c, ind_h, ind_w, dsdn, dsdc, dsdh, dsdw);
    

    // skip if not in foreground.
    if(dep_at_pix < 0.0f){
      continue;
    }

    // 相机距离张量cam_dist_in中，数据是按照一定顺序存储的。通过索引计算可以确定要访问的特定位置
    float cam_dist = cam_dist_in[n*cdisdn + ind_c*cdisdc]; // 获取 相机距离 张量在给定位置(n, ind_c)处的值
    float fl = fl_in[n*fisdn + ind_c*fisdc]; // 获取 焦距张量 在给定位置(n, ind_c)处的值

    // "高度索引的中心化值"是指将索引值转换为相对于张量高度中心的偏移量或位置，可以将像素或张量的位置相对于张量中心进行表示
    float imind_h = (float)ind_h - ((float)dszh-1.0f)/2.0f; // 计算像素高度索引ind_h的中心化值
    float imind_w = (float)ind_w - ((float)dszw-1.0f)/2.0f; // 计算像素宽度索引ind_w的中心化值
    
    // convert ray depth to plane depth
    float cos_theta = fl / vec3d_norm(imind_h, imind_w, fl);
    
    dep_at_pix = dep_at_pix * cos_theta;

    // find global coord
    // 全局坐标的原点是相机
    // 使用三角形相似的原理计算全局坐标
    float glob_y = -dep_at_pix*imind_w/fl; // 计算全局坐标中的y值
    float glob_z = -dep_at_pix*imind_h/fl; // 计算全局坐标中的z值
    float glob_x = dep_at_pix - cam_dist; // 计算全局坐标中的x值

    
    // find voxel index
    // 索引值表示了在体素数组中相应位置的体素
    int vox_ind_x = VOXIND_TO_VOXC(glob_x, vszx); // 将全局坐标glob_x映射到体素张量中的索引vox_ind_x
    int vox_ind_y = VOXIND_TO_VOXC(glob_y, vszy); // 将全局坐标glob_y映射到体素张量中的索引vox_ind_y
    int vox_ind_z = VOXIND_TO_VOXC(glob_z, vszz); // 将全局坐标glob_z映射到体素张量中的索引vox_ind_z

    
    // skip if out of bounds
    if(!WITHIN_BOUNDS(vox_ind_x, vox_ind_y, vox_ind_z, vszx, vszy, vszz)){
      continue;
    }
    

    // find voxel center
    // 首先需要将体素的索引值转换为相对于体素尺寸的比例值，将体素索引值加上0.5，并除以体素尺寸，这样可以将索引值映射到范围为 [0, 1] 的比例值
    // 接下来将范围为 [0, 1] 的比例值转换为范围为 [-0.5, 0.5] 的中心化值，从比例值中减去0.5，得到体素在 x 方向上的中心化值
    float vox_center_x = (((float)vox_ind_x+0.5f) / (float)vszx) - 0.5f; // 计算体素中心的x坐标
    float vox_center_y = (((float)vox_ind_y+0.5f) / (float)vszy) - 0.5f; // 计算体素中心的y坐标
    float vox_center_z = (((float)vox_ind_z+0.5f) / (float)vszz) - 0.5f; // 计算体素中心的z坐标

    
    //printf("%d %d %d \n", vox_ind_x, vox_ind_y, vox_ind_z);

    // calculate distance
    float dist = vec3d_norm(glob_x - vox_center_x, glob_y - vox_center_y, glob_z - vox_center_z);
    // 根据全局坐标和体素中心坐标，使用宏vec3d_norm计算距离dist

    /*if(vox_ind_x == 1 && vox_ind_y ==1 && vox_ind_z ==1) {
      printf("%f \n", dist - 1.0f/(float)(MAX3(vszx, vszy, vszz)));
      }*/

    // Assuming a tdf threshold of max cell length
    // 使用宏ADD_ATOMIC_5d对计数张量中给定索引处的值进行原子加法操作，将计数值增加
    ADD_ATOMIC_5d(voxel, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, vsdn, vsdc, vsdx, vsdy, vsdz, dist);
    ADD_ATOMIC_5d(cnt, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, csdn, csdc, csdx, csdy, csdz, 1.0f);
  }
}




__launch_bounds__(CUDA_NUM_THREADS)
__global__ void inplace_safe_divide( float* voxel,
                                     int N, int NC,
                                     int vszx, int vszy, int vszz,
                                     int vsdn, int vsdc, int vsdx, int vsdy, int vsdz,
                                     float* cnt,
                                     int csdn, int csdc,int csdx, int csdy, int csdz,
                                     float dist_bias,
                                     int nthreads){
  //int index;
  CUDA_KERNEL_LOOP(index, nthreads){
    //printf("hi!!!");
    const int n = index % N;
    const int ind_c = (index/ (N) ) % NC;
    const int ind_x = (index / (N * NC) ) % vszx;
    const int ind_y = (index / (N * NC * vszx) ) % vszy;
    const int ind_z = (index / (N * NC * vszx * vszy) ) % vszz;
    float ptnum = GET_DIRECT_5d(cnt, n, ind_c, ind_x, ind_y, ind_z, vsdn, vsdc, vsdx, vsdy, vsdz);
    if(ptnum < EPS){
      // ignore small values
      continue;
    }
    float dist = GET_DIRECT_5d(voxel, n, ind_c, ind_x, ind_y, ind_z, vsdn, vsdc, vsdx, vsdy, vsdz);
    SET_DIRECT_5d(voxel, n, ind_c, ind_x, ind_y, ind_z, csdn, csdc, csdx, csdy, csdz, (dist - dist_bias/(float)(MAX3(vszx, vszy, vszz)))/ptnum);
  }
}


__launch_bounds__(CUDA_NUM_THREADS)
__global__ void get_surface_mask_kernel(float* depth,
                                        int N, int NC,
                                        int dszh, int dszw,
                                        int dsdn, int dsdc, int dsdh, int dsdw,
                                        float* cam_dist_in,
                                        int cdisdn, int cdisdc,
                                        float* fl_in,
                                        int fisdn, int fisdc,
                                        float* cnt,
                                        int vszx, int vszy, int vszz,
                                        int csdn, int csdc, int csdx, int csdy, int csdz,
                                        float* mask,
                                        int msdn, int msdc, int msdx, int msdy, int msdz,
                                        int nthreads){
  CUDA_KERNEL_LOOP(index, nthreads){
    const int n = index % N;
    const int ind_c = (index / N) % NC;
    const int ind_x = (index / (N * NC)) % vszx;
    const int ind_y = (index / (N * NC * vszx) ) % vszy;
    const int ind_z = (index / (N * NC * vszx * vszy) ) % vszz;
    float fl = fl_in[n*fisdn+ind_c*fisdc];
    float cam_dist = cam_dist_in[n*cdisdn+ind_c*cdisdc];
    float ptnum = GET_DIRECT_5d(cnt, n, ind_c, ind_x, ind_y, ind_z, csdn, csdc, csdx, csdy, csdz);
    if(ptnum>EPS){
      continue;
    }
    float vox_center_x = (((float)(ind_x)+0.5)/(float)vszx) - 0.5;
    float vox_center_y = (((float)(ind_y)+0.5)/(float)vszy) - 0.5;
    float vox_center_z = (((float)(ind_z)+0.5)/(float)vszz) - 0.5;
    float im_h = -vox_center_z * fl/(vox_center_x + cam_dist);
    float im_w = -vox_center_y * fl/(vox_center_x + cam_dist);
    int im_idh = ROUND_I( 0.5*((float)dszh-1.0) + im_h );
    int im_idw = ROUND_I( 0.5*((float)dszw-1.0) + im_w );
    if(im_idh<0 || im_idh>=dszh){
      continue;
    }
    if(im_idw<0 || im_idw>=dszw){
      continue;
    }
    float dep_at_pix = GET_DIRECT_4d(depth, n, ind_c, im_idh, im_idw, dsdn, dsdc, dsdh, dsdw);
    if(dep_at_pix<0){
      continue;
    }
    float ray_depth = vec3d_norm(vox_center_x+cam_dist,vox_center_y,vox_center_z);
    if(dep_at_pix<ray_depth){
      SET_DIRECT_5d(mask, n, ind_c, ind_x, ind_y, ind_z, msdn, msdc, msdx, msdy, msdz, 0.0);
    }
  }
}




 

__launch_bounds__(CUDA_NUM_THREADS)
__global__ void back_projection_backward_kernel(float* depth,
                                                int N, int NC,
                                                int dszh, int dszw,
                                                int dsdn, int dsdc, int dsdh, int dsdw,
                                                float* fl_in,
                                                int fisdn, int fisdc,
                                                float* camdist_in,
                                                int cidn, int cidc,
                                                float* cnt,
                                                int cszx, int cszy, int cszz,
                                                int csdn, int csdc, int csdx, int csdy, int csdz,
                                                float* grad_in,
                                                int giszx, int giszy, int giszz,
                                                int gisdn, int gisdc, int gisdx, int gisdy, int gisdz,
                                                float* grad_depth,
                                                int gdsdn, int gdsdc, int gdsdh, int gdsdw,
                                                float* grad_camdist,
                                                int gcsdn, int gcsdc,
                                                float* grad_fl,
                                                int gfsdn, int gfsdc,
                                                int nthreads){
  CUDA_KERNEL_LOOP(index, nthreads){
    const int n = index % N;
    const int ind_c = (index / N) % NC;
    const int ind_h = (index / (N * NC)) % dszh;
    const int ind_w = (index / (N * NC * dszh) ) % dszw;
    // printf("%d %d %d %d\n", n, ind_c, ind_h, ind_w);
    float dep_at_pix_i = GET_DIRECT_4d(depth, n, ind_c, ind_h, ind_w, dsdn, dsdc, dsdh, dsdw);

    // skip if not in foreground.
    if(dep_at_pix_i < 0.0f){
      continue;
    }

    float fl = fl_in[n*fisdn+ind_c*fisdc];
    float cam_dist = camdist_in[n*csdn + ind_c*csdc];
    float imind_h = float(ind_h) - float(dszh-1)/2.0f;
    float imind_w = float(ind_w) - float(dszw-1)/2.0f;
    
    // convert ray depth to plane depth
    float cos_theta = fl / vec3d_norm(imind_h, imind_w, fl);
    float dep_at_pix = dep_at_pix_i * cos_theta;

    // find global coord
    float glob_y = -dep_at_pix*imind_w/fl;
    float glob_z = -dep_at_pix*imind_h/fl;
    float glob_x = dep_at_pix - cam_dist;

    // find voxel index
    int vox_ind_x = VOXIND_TO_VOXC(glob_x, cszx);
    int vox_ind_y = VOXIND_TO_VOXC(glob_y, cszy);
    int vox_ind_z = VOXIND_TO_VOXC(glob_z, cszz);

    // skip if out of bounds
    if(!WITHIN_BOUNDS(vox_ind_x, vox_ind_y, vox_ind_z, cszx, cszy, cszz)){
        continue;
    }
    

    

    // find voxel center
    float vox_center_x = ((float(vox_ind_x)+0.5) / float(cszx)) - 0.5;
    float vox_center_y = ((float(vox_ind_y)+0.5) / float(cszy)) - 0.5;
    float vox_center_z = ((float(vox_ind_z)+0.5) / float(cszz)) - 0.5;

    float pt_vec_len = vec3d_norm(imind_h, imind_w, fl);
    if(pt_vec_len < 1e-5){
      pt_vec_len = 1e-5;
    }
    float pt_dirvec_x = - fl /  pt_vec_len;
    float pt_dirvec_y = imind_w /  pt_vec_len;
    float pt_dirvec_z = imind_h /  pt_vec_len;

    float pt_vc_vec_len = vec3d_norm(glob_x - vox_center_x, glob_y - vox_center_y, glob_z - vox_center_z);
    if(pt_vc_vec_len < 1e-5){
      pt_vc_vec_len = 1e-5;
    }
    float pt_vc_dirvec_x = (glob_x - vox_center_x) /  pt_vc_vec_len;
    float pt_vc_dirvec_y = (glob_y - vox_center_y) /  pt_vc_vec_len;
    float pt_vc_dirvec_z = (glob_z - vox_center_z) /  pt_vc_vec_len;

    float cos_theta_cc = (pt_dirvec_x * pt_vc_dirvec_x) + (pt_dirvec_y * pt_vc_dirvec_y) + (pt_dirvec_z * pt_vc_dirvec_z);
    float ptnum = GET_DIRECT_5d(cnt, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, csdn, csdc, csdx, csdy, csdz);
    if(ptnum < 1){
      ptnum = 1;
    }

    float gd_i = GET_DIRECT_5d(grad_in, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, gisdn, gisdc, gisdx, gisdy, gisdz);
    SET_DIRECT_4d(grad_depth, n, ind_c, ind_h, ind_w, gdsdn, gdsdc, gdsdh, gdsdw, -gd_i*cos_theta_cc/ptnum);

    // grad_fl

    float grad_fl_x = ((glob_x-vox_center_x)/pt_vc_vec_len) * (square(imind_w) + square(imind_h)) / (pt_vec_len*pt_vec_len*pt_vec_len) ;
    float grad_fl_y = ((glob_y-vox_center_y)/pt_vc_vec_len) * (imind_w * fl) / (pt_vec_len*pt_vec_len*pt_vec_len) ;
    float grad_fl_z = ((glob_z-vox_center_z)/pt_vc_vec_len) * (imind_h * fl) / (pt_vec_len*pt_vec_len*pt_vec_len) ;
    float grad_fl_i = (grad_fl_x + grad_fl_y + grad_fl_z) * gd_i * dep_at_pix_i / ptnum;

    atomicAdd(grad_fl+gfsdn*n + gfsdc*ind_c, grad_fl_i);


    // grad_cam_dist

    atomicAdd(grad_camdist+gcsdn*n+gcsdc*ind_c, -pt_vc_dirvec_x*gd_i/ptnum);
  }
}


__launch_bounds__(CUDA_NUM_THREADS)
__global__ void spherical_back_projection_forward_kernel(float* depth,
                                               int N, int NC,
                                               int dszh, int dszw,
                                               int dsdn, int dsdc, int dsdh, int dsdw,
                                               float* grid_in,
                                               int gisdn, int gisdc,int gisdh, int gisdw,int gisddim,
                                               float* voxel,
                                               int vszx, int vszy, int vszz,
                                               int vsdn, int vsdc, int vsdx, int vsdy, int vsdz,
                                               float* cnt,
                                               int csdn, int csdc, int csdx, int csdy, int csdz,
                                               int nthreads){
  // int index;
  CUDA_KERNEL_LOOP(index, nthreads){
    const int n = index % N;
    const int ind_c = (index / N) % NC;
    const int ind_w = (index / (N * NC)) % dszh;
    const int ind_h = (index / (N * NC * dszh)) % dszw;
    
    float dep_at_pix = GET_DIRECT_4d(depth, n, ind_c, ind_h, ind_w, dsdn, dsdc, dsdh, dsdw);
    float grid_x = GET_DIRECT_5d(grid_in, n, ind_c, ind_h, ind_w, 0, gisdn, gisdc, gisdh, gisdw,gisddim);
    float grid_y = GET_DIRECT_5d(grid_in, n, ind_c, ind_h, ind_w, 1, gisdn, gisdc, gisdh, gisdw,gisddim);
    float grid_z = GET_DIRECT_5d(grid_in, n, ind_c, ind_h, ind_w, 2, gisdn, gisdc, gisdh, gisdw,gisddim);
    

    // skip if not in foreground.
    if(dep_at_pix < 0.0f){
      continue;
    }


    float glob_x = grid_x * dep_at_pix;
    float glob_y = grid_y * dep_at_pix;
    float glob_z = grid_z * dep_at_pix;

    
    // find voxel index
    int vox_ind_x = VOXIND_TO_VOXC(glob_x, vszx);
    int vox_ind_y = VOXIND_TO_VOXC(glob_y, vszy);
    int vox_ind_z = VOXIND_TO_VOXC(glob_z, vszz);

    
    // skip if out of bounds
    if(!WITHIN_BOUNDS(vox_ind_x, vox_ind_y, vox_ind_z, vszx, vszy, vszz)){
      continue;
    }
    

    // find voxel center
    float vox_center_x = (((float)vox_ind_x+0.5f) / (float)vszx) - 0.5f;
    float vox_center_y = (((float)vox_ind_y+0.5f) / (float)vszy) - 0.5f;
    float vox_center_z = (((float)vox_ind_z+0.5f) / (float)vszz) - 0.5f;

    
    //printf("%d %d %d \n", vox_ind_x, vox_ind_y, vox_ind_z);

    // calculate distance
    float dist = vec3d_norm(glob_x - vox_center_x, glob_y - vox_center_y, glob_z - vox_center_z);

    /*if(vox_ind_x == 1 && vox_ind_y ==1 && vox_ind_z ==1) {
      printf("%f \n", dist - 1.0f/(float)(MAX3(vszx, vszy, vszz)));
      }*/

    // Assuming a tdf threshold of max cell length
    ADD_ATOMIC_5d(voxel, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, vsdn, vsdc, vsdx, vsdy, vsdz, dist);
    ADD_ATOMIC_5d(cnt, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, csdn, csdc, csdx, csdy, csdz, 1.0f);
  }
}

__launch_bounds__(CUDA_NUM_THREADS)
__global__ void spherical_back_projection_backward_kernel(float* depth,
                                                int N, int NC,
                                                int dszh, int dszw,
                                                int dsdn, int dsdc, int dsdh, int dsdw,
                                                float* grid_in,
                                                int gisdn, int gisdc,int gisdh, int gisdw,int gisddim,
                                                float* cnt,
                                                int cszx, int cszy, int cszz,
                                                int csdn, int csdc, int csdx, int csdy, int csdz,
                                                float* grad_in,
                                                int giszx, int giszy, int giszz,
                                                int gradsdn, int gradsdc, int gradsdx, int gradsdy, int gradsdz,
                                                float* grad_depth,
                                                int gdsdn, int gdsdc, int gdsdh, int gdsdw,
                                                int nthreads){
  CUDA_KERNEL_LOOP(index, nthreads){

    const int n = index % N;
    const int ind_c = (index / N) % NC;
    const int ind_h = (index / (N * NC)) % dszh;
    const int ind_w = (index / (N * NC * dszh) ) % dszw;
    //printf("test");
    float dep_at_pix_i = GET_DIRECT_4d(depth, n, ind_c, ind_h, ind_w, dsdn, dsdc, dsdh, dsdw);
    float grid_x = GET_DIRECT_5d(grid_in, n, ind_c, ind_h, ind_w, 0, gisdn, gisdc, gisdh, gisdw,gisddim);
    float grid_y = GET_DIRECT_5d(grid_in, n, ind_c, ind_h, ind_w, 1, gisdn, gisdc, gisdh, gisdw,gisddim);
    float grid_z = GET_DIRECT_5d(grid_in, n, ind_c, ind_h, ind_w, 2, gisdn, gisdc, gisdh, gisdw,gisddim);
    
    

    // skip if not in foreground.
    if(dep_at_pix_i < 0.0f){
      continue;
    }
    float glob_x = grid_x * dep_at_pix_i;
    float glob_y = grid_y * dep_at_pix_i;
    float glob_z = grid_z * dep_at_pix_i;

    // find voxel index
    int vox_ind_x = VOXIND_TO_VOXC(glob_x, cszx);
    int vox_ind_y = VOXIND_TO_VOXC(glob_y, cszy);
    int vox_ind_z = VOXIND_TO_VOXC(glob_z, cszz);
    
    // skip if out of bounds
    if(!WITHIN_BOUNDS(vox_ind_x, vox_ind_y, vox_ind_z, cszx, cszy, cszz)){
        continue;
    }
    

    

    // find voxel center
    float vox_center_x = ((float(vox_ind_x)+0.5) / float(cszx)) - 0.5;
    float vox_center_y = ((float(vox_ind_y)+0.5) / float(cszy)) - 0.5;
    float vox_center_z = ((float(vox_ind_z)+0.5) / float(cszz)) - 0.5;

    float pt_vec_len = vec3d_norm(glob_x, glob_y, glob_z);
    if(pt_vec_len < 1e-5){
      pt_vec_len = 1e-5;
    }
    float pt_dirvec_x = glob_x /  pt_vec_len;
    float pt_dirvec_y = glob_y /  pt_vec_len;
    float pt_dirvec_z = glob_z /  pt_vec_len;

    float cos_theta_cc = (pt_dirvec_x * vox_center_x) + (pt_dirvec_y * vox_center_y) + (pt_dirvec_z * vox_center_z);
    float dist = vec3d_norm(glob_x - vox_center_x, glob_y - vox_center_y, glob_z - vox_center_z);

    float ptnum = GET_DIRECT_5d(cnt, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, csdn, csdc, csdx, csdy, csdz);
    // pts may be in diffrent voxel if its really near some places.
    //if( ptnum > 1-1e-5){
    if(ptnum < 1){
      ptnum = 1;
    }
    if(dist < 1e-5){
      dist=1e-5;
    }
    float gd_i = GET_DIRECT_5d(grad_in, n, ind_c, vox_ind_x, vox_ind_y, vox_ind_z, gradsdn, gradsdc, gradsdx, gradsdy, gradsdz);
    SET_DIRECT_4d(grad_depth, n, ind_c, ind_h, ind_w, gdsdn, gdsdc, gdsdh, gdsdw, gd_i*(dep_at_pix_i - cos_theta_cc)/(ptnum*dist));
      //}
      //else{
      //SET_DIRECT_4d(grad_depth, n, ind_c, ind_h, ind_w, gdsdn, gdsdc, gdsdh, gdsdw, 0);
      //}
  }
}

int spherical_back_proj_forward_wrap(THCState* state, THCudaTensor* depth, THCudaTensor* grid_in, THCudaTensor* voxel, THCudaTensor* cnt){
  
  THCUNN_assertSameGPU(state, 4, depth, grid_in, voxel, cnt);
  cambp_shapecheck(state, depth,grid_in, voxel, cnt);
  int N = THCudaTensor_size(state, depth, 0);
  int NC = THCudaTensor_size(state, depth, 1);
  int dszh = THCudaTensor_size(state, depth, 2);
  int dszw = THCudaTensor_size(state, depth, 3);
  int vszx = THCudaTensor_size(state, voxel, 2);
  int vszy = THCudaTensor_size(state, voxel, 3);
  int vszz = THCudaTensor_size(state, voxel, 4);
  THCudaTensor_resize5d(state, voxel, N, NC, vszx, vszy, vszz);
  THCudaTensor_resize5d(state, cnt, N, NC, vszx, vszy, vszz);
  THCudaTensor_zero(state, cnt);
  int count_im = (N * NC* dszh * dszw);
  int count_vox = (N*NC*vszx*vszy*vszz);
  
  spherical_back_projection_forward_kernel
    <<<GET_BLOCKS(count_im), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    THCudaTensor_data(state, depth),
    N,NC, dszh, dszw,
    THCudaTensor_stride(state, depth, 0),
    THCudaTensor_stride(state, depth, 1),
    THCudaTensor_stride(state, depth, 2),
    THCudaTensor_stride(state, depth, 3),
    THCudaTensor_data(state,grid_in),
    THCudaTensor_stride(state, grid_in, 0),
    THCudaTensor_stride(state, grid_in, 1),
    THCudaTensor_stride(state, grid_in, 2),
    THCudaTensor_stride(state, grid_in, 3),
    THCudaTensor_stride(state, grid_in, 4),
    THCudaTensor_data(state,voxel),
    vszx, vszy, vszz,
    THCudaTensor_stride(state, voxel, 0),
    THCudaTensor_stride(state, voxel, 1),
    THCudaTensor_stride(state, voxel, 2),
    THCudaTensor_stride(state, voxel, 3),
    THCudaTensor_stride(state, voxel, 4),
    THCudaTensor_data(state,cnt),
    THCudaTensor_stride(state, cnt, 0),
    THCudaTensor_stride(state, cnt, 1),
    THCudaTensor_stride(state, cnt, 2),
    THCudaTensor_stride(state, cnt, 3),
    THCudaTensor_stride(state, cnt, 4),
    count_im);
  hipError_t err = hipGetLastError();
 if (err != hipSuccess) {
    printf("error in projection foward: %s\n", hipGetErrorString(err));
    return 0;
  }
 //fprintf(stderr,"calling divide\n");
  inplace_safe_divide
    <<<GET_BLOCKS(count_vox), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    THCudaTensor_data(state, voxel),
    N, NC, vszx, vszy, vszz,
    THCudaTensor_stride(state, voxel, 0),
    THCudaTensor_stride(state, voxel, 1),
    THCudaTensor_stride(state, voxel, 2),
    THCudaTensor_stride(state, voxel, 3),
    THCudaTensor_stride(state, voxel, 4),
    THCudaTensor_data(state, cnt),
    THCudaTensor_stride(state, cnt, 0),
    THCudaTensor_stride(state, cnt, 1),
    THCudaTensor_stride(state, cnt, 2),
    THCudaTensor_stride(state, cnt, 3),
    THCudaTensor_stride(state, cnt, 4),
    0.0f,
    count_vox);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in inplace safe divide: %s\n", hipGetErrorString(err));
    return 0;
  }
  return 1;
}
int spherical_back_proj_backward_wrap(THCState* state, THCudaTensor* depth, THCudaTensor* grid_in, THCudaTensor* cnt, THCudaTensor* grad_in, THCudaTensor* grad_depth){
  THCUNN_assertSameGPU(state, 5, depth, grid_in, cnt, grad_in, grad_depth);
  
  int N = THCudaTensor_size(state, depth, 0);
  int NC = THCudaTensor_size(state, depth, 1);
  int dszh = THCudaTensor_size(state, depth, 2);
  int dszw = THCudaTensor_size(state, depth, 3);
  int vszx = THCudaTensor_size(state, cnt, 2);
  int vszy = THCudaTensor_size(state, cnt, 3);
  int vszz = THCudaTensor_size(state, cnt, 4);
  int count_im = (N * NC* dszh * dszw);
  
  spherical_back_projection_backward_kernel
    <<<GET_BLOCKS(count_im), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    THCudaTensor_data(state, depth),
    N,NC, dszh, dszw,
    THCudaTensor_stride(state, depth, 0),
    THCudaTensor_stride(state, depth, 1),
    THCudaTensor_stride(state, depth, 2),
    THCudaTensor_stride(state, depth, 3),
    THCudaTensor_data(state,grid_in),
    THCudaTensor_stride(state, grid_in, 0),
    THCudaTensor_stride(state, grid_in, 1),
    THCudaTensor_stride(state, grid_in, 2),
    THCudaTensor_stride(state, grid_in, 3),
    THCudaTensor_stride(state, grid_in, 4),
    THCudaTensor_data(state,cnt),
    vszx, vszy, vszz,
    THCudaTensor_stride(state, cnt, 0),
    THCudaTensor_stride(state, cnt, 1),
    THCudaTensor_stride(state, cnt, 2),
    THCudaTensor_stride(state, cnt, 3),
    THCudaTensor_stride(state, cnt, 4),
    THCudaTensor_data(state, grad_in),
    vszx, vszy, vszz,
    THCudaTensor_stride(state, grad_in, 0),
    THCudaTensor_stride(state, grad_in, 1),
    THCudaTensor_stride(state, grad_in, 2),
    THCudaTensor_stride(state, grad_in, 3),
    THCudaTensor_stride(state, grad_in, 4),
    THCudaTensor_data(state, grad_depth),
    THCudaTensor_stride(state, grad_depth, 0),
    THCudaTensor_stride(state, grad_depth, 1),
    THCudaTensor_stride(state, grad_depth, 2),
    THCudaTensor_stride(state, grad_depth, 3),
    count_im);
  
  hipError_t err = hipGetLastError();
 if (err != hipSuccess) {
    printf("error in projection foward: %s\n", hipGetErrorString(err));
    return 0;
  }
  return 1;
}


int back_projection_forward_wrap(THCState* state, THCudaTensor* depth, THCudaTensor* camdist, THCudaTensor* fl, THCudaTensor* voxel, THCudaTensor* cnt){
  //fprintf(stderr,"calling cuda!!\n");
  THCUNN_assertSameGPU(state, 5, depth, camdist, fl, voxel, cnt);
  cambp_shapecheck(state, depth, camdist, fl, voxel, cnt);
  int N = THCudaTensor_size(state, depth, 0);
  int NC = THCudaTensor_size(state, depth, 1);
  int dszh = THCudaTensor_size(state, depth, 2);
  int dszw = THCudaTensor_size(state, depth, 3);
  int vszx = THCudaTensor_size(state, voxel, 2);
  int vszy = THCudaTensor_size(state, voxel, 3);
  int vszz = THCudaTensor_size(state, voxel, 4);
  THCudaTensor_resize5d(state, voxel, N, NC, vszx, vszy, vszz);
  THCudaTensor_resize5d(state, cnt, N, NC, vszx, vszy, vszz);
  THCudaTensor_zero(state, cnt);
  int count_im = (N * NC* dszh * dszw);
  int count_vox = (N*NC*vszx*vszy*vszz);
  //fprintf(stderr,"calling forawrd\n");

 back_projection_forward_kernel
    <<<GET_BLOCKS(count_im), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    THCudaTensor_data(state, depth),
    N,NC, dszh, dszw,
    THCudaTensor_stride(state, depth, 0),
    THCudaTensor_stride(state, depth, 1),
    THCudaTensor_stride(state, depth, 2),
    THCudaTensor_stride(state, depth, 3),
    THCudaTensor_data(state, camdist),
    THCudaTensor_stride(state,camdist, 0),
    THCudaTensor_stride(state,camdist, 1),
    THCudaTensor_data(state,fl),
    THCudaTensor_stride(state, fl, 0),
    THCudaTensor_stride(state, fl, 1),
    THCudaTensor_data(state,voxel),
    vszx, vszy, vszz,
    THCudaTensor_stride(state, voxel, 0),
    THCudaTensor_stride(state, voxel, 1),
    THCudaTensor_stride(state, voxel, 2),
    THCudaTensor_stride(state, voxel, 3),
    THCudaTensor_stride(state, voxel, 4),
    THCudaTensor_data(state,cnt),
    THCudaTensor_stride(state, cnt, 0),
    THCudaTensor_stride(state, cnt, 1),
    THCudaTensor_stride(state, cnt, 2),
    THCudaTensor_stride(state, cnt, 3),
    THCudaTensor_stride(state, cnt, 4),
    count_im);
 hipError_t err = hipGetLastError();
 if (err != hipSuccess) {
    printf("error in projection foward: %s\n", hipGetErrorString(err));
    return 0;
  }

 //fprintf(stderr,"calling divide\n");

  inplace_safe_divide
    <<<GET_BLOCKS(count_vox), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    THCudaTensor_data(state, voxel),
    N, NC, vszx, vszy, vszz,
    THCudaTensor_stride(state, voxel, 0),
    THCudaTensor_stride(state, voxel, 1),
    THCudaTensor_stride(state, voxel, 2),
    THCudaTensor_stride(state, voxel, 3),
    THCudaTensor_stride(state, voxel, 4),
    THCudaTensor_data(state, cnt),
    THCudaTensor_stride(state, cnt, 0),
    THCudaTensor_stride(state, cnt, 1),
    THCudaTensor_stride(state, cnt, 2),
    THCudaTensor_stride(state, cnt, 3),
    THCudaTensor_stride(state, cnt, 4),
    1.0f,
    count_vox);

  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in inplace safe divide: %s\n", hipGetErrorString(err));
    return 0;
  }
  return 1;
}

int get_surface_mask_wrap(THCState* state, THCudaTensor* depth, THCudaTensor* camdist, THCudaTensor* fl, THCudaTensor* cnt, THCudaTensor* mask){
  
  THCUNN_assertSameGPU(state, 5, depth, camdist, fl, mask, cnt);
  cambp_shapecheck(state, depth, camdist, fl, mask, cnt);
  int N = THCudaTensor_size(state, depth, 0);
  int NC = THCudaTensor_size(state, depth, 1);
  int dszh = THCudaTensor_size(state, depth, 2);
  int dszw = THCudaTensor_size(state, depth, 3);
  int vszx = THCudaTensor_size(state, mask, 2);
  int vszy = THCudaTensor_size(state, mask, 3);
  int vszz = THCudaTensor_size(state, mask, 4);
  THCudaTensor_resize5d(state, mask, N, NC, vszx, vszy, vszz);
  THCudaTensor_resize5d(state, cnt, N, NC, vszx, vszy, vszz);
  THCudaTensor_fill(state, mask, 1.0);
  
  int count_vox = (N*NC*vszx*vszy*vszz);
  //fprintf(stderr,"calling forawrd\n");
 get_surface_mask_kernel
    <<<GET_BLOCKS(count_vox), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
    THCudaTensor_data(state, depth),
    N, NC, dszh, dszw,
    THCudaTensor_stride(state, depth, 0),
    THCudaTensor_stride(state, depth, 1),
    THCudaTensor_stride(state, depth, 2),
    THCudaTensor_stride(state, depth, 3),
    THCudaTensor_data(state, camdist),
    THCudaTensor_stride(state,camdist, 0),
    THCudaTensor_stride(state,camdist, 1),
    THCudaTensor_data(state,fl),
    THCudaTensor_stride(state, fl, 0),
    THCudaTensor_stride(state, fl, 1),
    THCudaTensor_data(state,cnt),
    vszx, vszy, vszz,
    THCudaTensor_stride(state,cnt, 0),
    THCudaTensor_stride(state, cnt, 1),
    THCudaTensor_stride(state, cnt, 2),
    THCudaTensor_stride(state, cnt, 3),
    THCudaTensor_stride(state, cnt, 4),
    THCudaTensor_data(state,mask),
    THCudaTensor_stride(state, mask, 0),
    THCudaTensor_stride(state, mask, 1),
    THCudaTensor_stride(state, mask, 2),
    THCudaTensor_stride(state, mask, 3),
    THCudaTensor_stride(state, mask, 4),
    count_vox);
 hipError_t err = hipGetLastError();
 if (err != hipSuccess) {
    printf("error in projection foward: %s\n", hipGetErrorString(err));
    return 0;
  }
 return 1;
}



//backward

int back_projection_backward_wrap (THCState* state, THCudaTensor* depth, THCudaTensor* fl, THCudaTensor* camdist, THCudaTensor* cnt, THCudaTensor* grad_in, THCudaTensor* grad_depth, THCudaTensor* grad_camdist, THCudaTensor* grad_fl ){
  THCUNN_assertSameGPU(state, 7, depth, fl, cnt, grad_in, grad_depth, grad_camdist, grad_fl);
  int N = THCudaTensor_size(state, depth, 0);
  int NC = THCudaTensor_size(state, depth, 1);
  int dszh = THCudaTensor_size(state, depth, 2);
  int dszw = THCudaTensor_size(state, depth, 3);
  int cszx = THCudaTensor_size(state, cnt, 2);
  int cszy = THCudaTensor_size(state, cnt, 3);
  int cszz = THCudaTensor_size(state, cnt, 4);
  THCudaTensor_resize4d(state, grad_depth, N, NC, dszh, dszw);
  THCudaTensor_resize2d(state, grad_camdist, N, NC);
  THCudaTensor_resize2d(state, grad_fl, N, NC);
  THCudaTensor_zero(state, grad_depth);
  THCudaTensor_zero(state, grad_camdist);
  THCudaTensor_zero(state, grad_fl);
  int count_im = (N * NC* dszh * dszw);

  back_projection_backward_kernel
    <<<GET_BLOCKS(count_im), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
                                                                                   THCudaTensor_data(state, depth),
                                                                                   N, NC,
                                                                                   dszh, dszw,
                                                                                   THCudaTensor_stride(state, depth, 0),
                                                                                   THCudaTensor_stride(state, depth, 1),
                                                                                   THCudaTensor_stride(state, depth, 2),
                                                                                   THCudaTensor_stride(state, depth, 3),
                                                                                   THCudaTensor_data(state, fl),
                                                                                   THCudaTensor_stride(state, fl, 0),
                                                                                   THCudaTensor_stride(state, fl, 1),
                                                                                   THCudaTensor_data(state, camdist),
                                                                                   THCudaTensor_stride(state, camdist, 0),
                                                                                   THCudaTensor_stride(state, camdist, 1),
                                                                                   THCudaTensor_data(state, cnt),
                                                                                   cszx, cszy, cszz,
                                                                                   THCudaTensor_stride(state, cnt, 0),
                                                                                   THCudaTensor_stride(state, cnt, 1),
                                                                                   THCudaTensor_stride(state, cnt, 2),
                                                                                   THCudaTensor_stride(state, cnt, 3),
                                                                                   THCudaTensor_stride(state, cnt, 4),
                                                                                   THCudaTensor_data(state, grad_in),
                                                                                   cszx,cszy,cszz,
                                                                                   THCudaTensor_stride(state, grad_in, 0),
                                                                                   THCudaTensor_stride(state, grad_in, 1),
                                                                                   THCudaTensor_stride(state, grad_in, 2),
                                                                                   THCudaTensor_stride(state, grad_in, 3),
                                                                                   THCudaTensor_stride(state, grad_in, 4),
                                                                                   THCudaTensor_data(state, grad_depth),
                                                                                   THCudaTensor_stride(state, grad_depth, 0),
                                                                                   THCudaTensor_stride(state, grad_depth, 1),
                                                                                   THCudaTensor_stride(state, grad_depth, 2),
                                                                                   THCudaTensor_stride(state, grad_depth, 3),
                                                                                   THCudaTensor_data(state, grad_camdist),
                                                                                   THCudaTensor_stride(state,grad_camdist, 0),
                                                                                   THCudaTensor_stride(state,grad_camdist, 1),
                                                                                   THCudaTensor_data(state, grad_fl),
                                                                                   THCudaTensor_stride(state,grad_fl, 0),
                                                                                   THCudaTensor_stride(state,grad_fl, 1),
                                                                                   count_im);

                    
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in BilinearSampler3D update gradInput: %s\n", hipGetErrorString(err));
    return 0;
  }
  return 1;
 }
 
